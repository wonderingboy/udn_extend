#include "hip/hip_runtime.h"
#include <vector>
#include "caffe/util/math_functions.hpp"
#include "caffe/common_layers.hpp"
using std::max;
using std::min;
using std::floor;

namespace caffe {

// Copy (one line per thread) from one array to another, with arbitrary
// strides in the last two dimensions.
template <typename Dtype>
__global__ void copy_kernel(const int nthreads, const int x_center, const int y_center,
    const int candidate_width, const int candidate_height, const int search_width, const int search_height, const int channels, const int halfwidth, const int halfheight,   
    const Dtype* src, Dtype* dest) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int w = index % candidate_width;
    int h = (index / candidate_width)%candidate_height;
    int c = (index / candidate_width / candidate_height) % channels;
    int n = index / candidate_width / candidate_height / channels;

    int new_w_c = x_center + w - halfwidth;
    int new_h_c = y_center + h - halfheight;
	if (new_w_c < 0 || new_w_c >= search_width || new_h_c < 0 || new_h_c >= search_height){
		dest[index] = 0; 
	}else{        
		dest[index] = src[n * channels*search_height*search_width + c * search_height * search_width + new_h_c * search_width + new_w_c];
	}	    
  }
}

// Back the a to b
// src is a and the the dest b
template <typename Dtype>
__global__ void back_kernel(const int nthreads, const int x_center, const int y_center,
    const int candidate_width, const int candidate_height, const int search_width, const int search_height, const int channels, const int halfwidth, const int halfheight,
    const Dtype* src, Dtype* dest) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int w = index % candidate_width;
    int h = (index / candidate_width)%candidate_height;
    int c = (index / candidate_width / candidate_height) % channels;
    int n = index / candidate_width / candidate_height / channels;

    int new_w_c = x_center + w - halfwidth;
    int new_h_c = y_center + h - halfheight;

    int dest_index = n * channels*search_height*search_width + c * search_height * search_width + new_h_c * search_width + new_w_c;
    if (new_w_c < 0 || new_w_c >= search_width || new_h_c < 0 || new_h_c >= search_height){
         dest[dest_index] = dest[dest_index];
    }else{
         dest[dest_index] = dest[dest_index] + src[index];
    }
  }
}
 
template <typename Dtype>
void CorrelationLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {

  const Dtype* bottom_candidate = bottom[0]->gpu_data(); 
  const Dtype* bottom_search = bottom[1]->gpu_data();
  Dtype* sub_region_data = sub_region.mutable_gpu_data(); 
  //const Dtype* sub_region_val = sub_region.cpu_data();


  int candidate_width = bottom[0]->width();
  int candidate_height = bottom[0]->height();  
  int channels = bottom[0]->channels();
  int num = bottom[0]->num();
  int lines = bottom[0]->count()/num;

  int search_width = bottom[1]->width();
  int search_height = bottom[1]->height();
  int halfwidth = floor(candidate_width/2);  
  int halfheight = floor(candidate_height/2);

//  top[0]->Reshape(num, 1, search_width, search_height);
//  while(1);
  //LOG(INFO)<<top[0]->num()<<top[0]->channels();
  //
  Dtype* top_data = top[0]->mutable_cpu_data();
  for(int i=0; i<num; i++){
      Dtype* top_data_index = top_data + top[0]->offset(i); 
      const Dtype* bottom_candidate_index = bottom_candidate + bottom[0]->offset(i);
      const Dtype* bottom_search_index = bottom_search + bottom[1]->offset(i);
      
      Dtype bottom_candidate_norm = 0;
      caffe_gpu_dot(lines, bottom_candidate_index, bottom_candidate_index, &bottom_candidate_norm);
     // LOG(INFO)<< bottom_search_index;
      for(int x_start = 0; x_start < search_width; ++x_start){
	for(int y_start = 0; y_start < search_height; ++y_start){
            Dtype rst = 0;
            //LOG(INFO)<<i<<",x: "<<x_start<<",y: "<<y_start;
	    copy_kernel<Dtype><<<CAFFE_GET_BLOCKS(lines), CAFFE_CUDA_NUM_THREADS>>>(lines, x_start, y_start, candidate_width, candidate_height, search_width, search_height, channels, halfwidth, halfheight, bottom_search_index, sub_region_data);
            caffe_gpu_dot(lines, sub_region_data, bottom_candidate_index, &rst);

            Dtype bottom_search_norm = 0;
            caffe_gpu_dot(lines, sub_region_data, sub_region_data, &bottom_search_norm);

/*	    if (x_start == 10 && y_start == 10){ 
              Dtype* sub_region_val = sub_region.mutable_cpu_data();
              for(int layer_count = 2; layer_count <3; layer_count++){ 
               	for(int sub_i=0; sub_i<7; ++sub_i){
		    for(int sub_j=0; sub_j<7; ++sub_j){
		        LOG(INFO)<<sub_i<<","<<sub_j<<": "<<sub_region_val[layer_count*49 + sub_i*7+sub_j];
	            }
                }
              }
              LOG(INFO)<<y_start*search_width + x_start<<": "<<rst;
            }*/
            //LOG(INFO)<<y_start*search_width + x_start<<": "<<rst;
            top_data_index[y_start*search_width + x_start] = rst / sqrt(bottom_candidate_norm + 1) / sqrt(bottom_search_norm + 1);
	}
      }
  }
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
void CorrelationLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* top_diff = top[0]->cpu_diff();
  const Dtype* bottom_candidate = bottom[0]->gpu_data();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  Dtype* bottom_diff_big = bottom[1]->mutable_gpu_diff();
  Dtype* sub_region_data = sub_region.mutable_gpu_data(); 
  const Dtype* bottom_search = bottom[1]->gpu_data();

  int num = bottom[0]->num();
  int channels = bottom[0]->channels();
  int candidate_width = bottom[0]->width();
  int candidate_height = bottom[0]->height();  
  int search_width = bottom[1]->width();
  int search_height = bottom[1]->height();
  
  int lines = bottom[0]->count()/num;
  int halfwidth = floor(candidate_width/2);
  int halfheight = floor(candidate_height/2);

//  LOG(INFO)<<"top_diff size:"<<top[0]->num()<<top[0]->channels();
  if (propagate_down[0]) {
       for(int i=0; i<num; ++i){
          const Dtype* top_diff_index = top_diff + top[0]->offset(i);  
          Dtype* bottom_candidate_diff_index = bottom_diff + bottom[0]->offset(i);
          Dtype* bottom_search_diff_index = bottom_diff_big + bottom[1]->offset(i);
          const Dtype* bottom_search_index = bottom_search + bottom[1]->offset(i);
                   
 
          const Dtype* bottom_candidate_index = bottom_candidate + bottom[0]->offset(i);
          Dtype bottom_candidate_norm = 0;
          caffe_gpu_dot(lines, bottom_candidate_index, bottom_candidate_index, &bottom_candidate_norm);

	  for(int x_start = 0; x_start < search_width; ++x_start){
		for(int y_start = 0; y_start < search_height; ++y_start){
			copy_kernel<Dtype><<<CAFFE_GET_BLOCKS(lines), CAFFE_CUDA_NUM_THREADS>>>(lines, x_start, y_start, candidate_width, candidate_height, search_width, search_height, channels,halfwidth,halfheight, bottom_search_index, sub_region_data);

                        Dtype bottom_search_norm = 0;
                        caffe_gpu_dot(lines, sub_region_data, sub_region_data, &bottom_search_norm);

			caffe_gpu_axpy(
			    lines,              // count
			    top_diff_index[y_start*search_width + x_start] / sqrt(bottom_candidate_norm + 1) / sqrt(bottom_search_norm + 1),                              // alpha
			    sub_region_data,                   // a			 
			    bottom_candidate_diff_index);  // b
                       // if(x_start == 10 && y_start == 10){
		       //     LOG(INFO)<<"diff: "<<y_start<<","<<x_start<<","<<top_diff_index[y_start*search_width+x_start];
			//}
                        // for the search diff
                       /*caffe_gpu_set(lines, Dtype(0), sub_region_data);
                       caffe_gpu_axpy(
                            lines,              // count
                            top_diff_index[y_start*search_width + x_start],                              // alpha
                            bottom[0]->gpu_data(),                   // a                      
                            sub_region_data);  
                       back_kernel<Dtype><<<CAFFE_GET_BLOCKS(lines), CAFFE_CUDA_NUM_THREADS>>>(lines, x_start, y_start, candidate_width, candidate_height, search_width, search_height, channels,halfwidth,halfheight, sub_region_data, bottom_search_diff_index);*/
		}
	  }
       }
  CUDA_POST_KERNEL_CHECK;
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(CorrelationLayer);

}  // namespace caffe
